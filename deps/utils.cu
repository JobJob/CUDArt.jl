
#include <hip/hip_runtime.h>
#include <stdint.h>
//#include <stdio.h>

template <typename T>
__device__ void fill_contiguous(T *data, size_t len, T val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = idx; i < len; i += gridDim.x * blockDim.x) {
        data[i] = val;
    }
}

// Note that pitch must be expressed in elements, not bytes!
template <typename T>
__device__ void fill_pitched(T *data, size_t width, size_t height, size_t depth, size_t pitch, T val) {
    int idxx = blockIdx.x * blockDim.x + threadIdx.x;
    int idxy = blockIdx.y * blockDim.y + threadIdx.y;
    int idxz = blockIdx.z * blockDim.z + threadIdx.z;

    for (int iz = idxz; iz < depth; iz += gridDim.z * blockDim.z) {
        int offsetz = iz;
        for (int iy = idxy; iy < height; iy += gridDim.y * blockDim.y) {
            int offsety = height*offsetz + iy;
            for (int ix = idxx; ix < width; ix += gridDim.x * blockDim.x) {
                data[pitch*offsety + ix] = val;
            }
        }
    }
}

// To build a library, we have to explicitly instantiate these.
// Don't wrap in extern "C", use Cpp.jl to do the name-mangling.
// template void fill_contiguous<double>(double *, size_t, double);
// template void fill_contiguous<float>(float *, size_t, float);
// template void fill_contiguous<int64_t>(int64_t *, size_t, int64_t);
// template void fill_contiguous<uint64_t>(uint64_t *, size_t, uint64_t);
// template void fill_contiguous<int32_t>(int32_t *, size_t, int32_t);
// template void fill_contiguous<uint32_t>(uint32_t *, size_t, uint32_t);
// template void fill_contiguous<int16_t>(int16_t *, size_t, int16_t);
// template void fill_contiguous<uint16_t>(uint16_t *, size_t, uint16_t);
// template void fill_contiguous<int8_t>(int8_t *, size_t, int8_t);
// template void fill_contiguous<uint8_t>(uint8_t *, size_t, uint8_t);
// 
// template void fill_pitched<double>(double*, size_t, size_t, size_t, size_t, double);
// template void fill_pitched<float>(float*, size_t, size_t, size_t, size_t, float);
// template void fill_pitched<int64_t>(int64_t*, size_t, size_t, size_t, size_t, int64_t);
// template void fill_pitched<uint64_t>(uint64_t*, size_t, size_t, size_t, size_t, uint64_t);
// template void fill_pitched<int32_t>(int32_t*, size_t, size_t, size_t, size_t, int32_t);
// template void fill_pitched<uint32_t>(uint32_t*, size_t, size_t, size_t, size_t, uint32_t);
// template void fill_pitched<int16_t>(int16_t*, size_t, size_t, size_t, size_t, int16_t);
// template void fill_pitched<uint16_t>(uint16_t*, size_t, size_t, size_t, size_t, uint16_t);
// template void fill_pitched<int8_t>(int8_t*, size_t, size_t, size_t, size_t, int8_t);
// template void fill_pitched<uint8_t>(uint8_t*, size_t, size_t, size_t, size_t, uint8_t);

extern "C"
{
    void __global__ fill_contiguous_double(double *data, size_t len, double val) {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_float(float *data, size_t len, float val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int64(int64_t *data, size_t len, int64_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint64(uint64_t *data, size_t len, uint64_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int32(int32_t *data, size_t len, int32_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint32(uint32_t *data, size_t len, uint32_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int16(int16_t *data, size_t len, int16_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint16(uint16_t *data, size_t len, uint16_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_int8(int8_t *data, size_t len, int8_t val)  {fill_contiguous(data, len, val);}
    void __global__ fill_contiguous_uint8(uint8_t *data, size_t len, uint8_t val)  {fill_contiguous(data, len, val);}

    void __global__ fill_pitched_double(double *data, size_t width, size_t height, size_t depth, size_t pitch, double val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_float(float *data, size_t width, size_t height, size_t depth, size_t pitch, float val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int64(int64_t *data, size_t width, size_t height, size_t depth, size_t pitch, int64_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint64(uint64_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint64_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int32(int32_t *data, size_t width, size_t height, size_t depth, size_t pitch, int32_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint32(uint32_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint32_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int16(int16_t *data, size_t width, size_t height, size_t depth, size_t pitch, int16_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint16(uint16_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint16_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_int8(int8_t *data, size_t width, size_t height, size_t depth, size_t pitch, int8_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
    void __global__ fill_pitched_uint8(uint8_t *data, size_t width, size_t height, size_t depth, size_t pitch, uint8_t val) {
        fill_pitched(data, width, height, depth, pitch, val);
    }
}

/*// Debugging
int main()
{
  size_t width = 5;
  size_t height = 3;
  size_t depth = 1;
  size_t pitch = sizeof(double)*width;
  cudaSetDevice(0);
  cudaExtent extent = make_cudaExtent(pitch, height, depth);
  cudaPitchedPtr devpp;
  cudaMalloc3D(&devpp, extent);
  pitch = devpp.pitch;
  fill_pitched_double<<<32,16>>>((double *) devpp.ptr, width, height, depth, pitch/sizeof(double), (double) 8.4);
  cudaDeviceSynchronize();
  double *host = new double[width*height];
  cudaPos dstpos = make_cudaPos(0, 0, 0);
  cudaPos srcpos = make_cudaPos(0, 0, 0);
  cudaPitchedPtr hostpp = make_cudaPitchedPtr(host, sizeof(double)*width, width, height);
  cudaMemcpy3DParms mcpp = {0};
  mcpp.srcPos = srcpos;
  mcpp.dstPos = dstpos;
  mcpp.srcPtr = devpp;
  mcpp.dstPtr = hostpp;
  mcpp.extent = extent;
  mcpp.kind = cudaMemcpyDeviceToHost;
  cudaMemcpy3D(&mcpp);
  cudaDeviceSynchronize();
  for (int i = 0; i < width*height; i++)
    printf("%g\n", host[i]);
  cudaDeviceReset();
}*/